#include "hip/hip_runtime.h"
#include <math.h>
#include <builtin_types.h>

#define INDEX_SIZE ${size}
#define BLOCK_SIZE ${block}
#define STEP_SIZE ${stepSize}

extern "C"
__global__ void calculateDistance(int *index, int *query, float *distances, unsigned int *validComps,
		unsigned int *seqs,
		unsigned int *indexIncrement,
		float scale, unsigned int numSeqs, unsigned int length, float sliceDistance);

extern "C"
__global__ void setToZero(float *comps);

extern "C"
__global__ void calculateQgrams(char *sequence, unsigned int q, unsigned int length, float *comps, float windowLength, float step, float fraction, char nAs);


__global__ void calculateDistance(int *index, int *query, float *distances, unsigned int *validComps,
		unsigned int *seqs,
		unsigned int *indexIncrement, float scale, unsigned int numSeqs, unsigned int length, float sliceDistance){


	unsigned int seq = blockIdx.y / BLOCK_SIZE;
	unsigned int blockY = blockIdx.y % BLOCK_SIZE;
	//unsigned int indexSeq = seq * STEP_SIZE;
	unsigned int comp = blockIdx.x * BLOCK_SIZE + blockY;
	unsigned int block = (comp)*(INDEX_SIZE+1);
	unsigned int threadPlus1 = threadIdx.x+1;
	unsigned int thread = threadIdx.x;

	if (comp < length) {
		__shared__ float s_distances[INDEX_SIZE];
		__shared__ int s_notEmptyQ[INDEX_SIZE];
		unsigned int indexValue = index[block+threadPlus1];
		unsigned int queryValue = query[threadPlus1+(seq*(INDEX_SIZE+1))];

		s_distances[thread] = (float) indexValue - (float)queryValue;
		s_distances[thread] *= s_distances[thread];

		s_notEmptyQ[thread] = indexValue | queryValue;

		unsigned int offset = 1;

		for (int i=INDEX_SIZE>>1; i > 0 ; i >>=1 ) {
			__syncthreads();
			if (thread < i) {
				int ai = offset*(2*thread+1)-1;
				int bi = offset*(2*thread+2)-1;
				s_distances[bi] += s_distances[ai];
				s_notEmptyQ[bi] |= s_notEmptyQ[ai];
			}
			offset *= 2;
		}
		__syncthreads();
		if (thread == 0 && s_notEmptyQ[INDEX_SIZE-1]){
			s_distances[INDEX_SIZE-1] = sqrt(s_distances[INDEX_SIZE-1])/scale;
			if (s_distances[INDEX_SIZE-1] < sliceDistance){
				unsigned int indices = atomicAdd(indexIncrement, 1);
				distances[indices] =  s_distances[INDEX_SIZE-1];
				validComps[indices] = comp;
				seqs[indices] = seq;
			}
		}

	}
}

__global__ void setToZero(float *comps){
	unsigned int index = 1+threadIdx.x + (INDEX_SIZE+1) * (blockIdx.x*BLOCK_SIZE + blockIdx.y);
	comps[index] = 0.0;
}

__global__ void calculateQgrams(char *sequence, unsigned int q, unsigned int length, float *comps, float windowLength, float step, float fraction, char nAs) {
	unsigned int seqLocation = threadIdx.x + INDEX_SIZE * blockIdx.x;
	if (seqLocation < length - q) {
		int localQgram = 0;
		int bit = 1;
		//for (int i=q-1; i >= 0; i--) {
		for (int i=0; i < q; i++) {
			if (localQgram >= 0) {
				char character = sequence[seqLocation+i];
				if (character == 'N')
					character = nAs;
				switch (character) {
					case 'A' : break;
					case 'T' : localQgram+=1*bit; break;
					case 'C' : localQgram+=2*bit; break;
					case 'G' : localQgram+=3*bit; break;
					default : localQgram = -1;
				}
				bit *= q;
			}
		}
		if (localQgram >= 0) {
			localQgram++;
			unsigned int startWindow = (seqLocation-windowLength) < 0 ? 0 : (unsigned int)ceil((seqLocation - windowLength) / step);
			unsigned int endWindow = (unsigned int)floor(seqLocation/step) < STEP_SIZE? (unsigned int)floor(seqLocation/step) : STEP_SIZE-1;
			comps[(startWindow * (INDEX_SIZE+1))]= (float)windowLength;

			for (unsigned int i =startWindow; i < endWindow; i++){
				atomicAdd(&comps[i * (INDEX_SIZE+1)+localQgram], fraction);
			}

		}

	}

}


